#include "hip/hip_runtime.h"
#include "image_set.cuh"
#include "tools/cuda_utils.cuh"
namespace photo {

  void ImageSet::getIntensity(Matrixf& intensity) const {
    intensity.resize(rows(), cols());
    intensity.fill(0);
    for (size_t k = 0; k < matrix_.size(); ++k)
      intensity.at(k) = matrix_.at(k).intensity();
  }

  void ImageSet::getDepth(Matrixf& depth) const {
    depth.resize(rows(), cols());
    depth.fill(0);
    for (size_t k = 0; k < matrix_.size(); ++k)
      depth.at(k) = matrix_.at(k).depth();
  }

  // TODO fix redundant arguments
  __global__ void toCloud_kernel(MatrixCloud* target,
                                 const Image* mat,
                                 const Eigen::Isometry3f sensor_offset,
                                 const Eigen::Matrix3f inv_K,
                                 const CameraType cam_type,
                                 const float ifx,
                                 const float ify,
                                 const float cx,
                                 const float cy,
                                 const float min_depth,
                                 const float max_depth) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (!target->inside(row, col))
      return;

    auto& dst            = target->at<1>(row, col);
    const ImageEntry src = mat->at<1>(row, col);

    float w = src.depth();
    if (src.masked() || w < min_depth || w > max_depth)
      return;
#ifdef _MD_ENABLE_SUPERRES_
    const float& r = src.r;
    const float& c = src.c;
#else
    const float r = row;
    const float c = col;
#endif
    dst.status = PointStatusFlag::Good;
    switch (cam_type) {
      case Pinhole: {
        dst.coordinates = inv_K * Eigen::Vector3f(c * w, r * w, w);
      } break;
      case Spherical: {
        float azimuth      = ifx * (c - cx);
        float elevation    = ify * (r - cy);
        float s0           = sinf(azimuth);
        float c0           = cosf(azimuth);
        float s1           = sinf(elevation);
        float c1           = cosf(elevation);
        dst.coordinates(0) = c0 * c1 * w;
        dst.coordinates(1) = s0 * c1 * w;
        dst.coordinates(2) = s1 * w;
      } break;
      default:;
    }
    dst.intensity   = src.intensity();
    dst.normal      = sensor_offset.linear() * src.normal();
    dst.coordinates = sensor_offset * dst.coordinates;
  }

  void ImageSet::toCloudDevice(MatrixCloud* target) const {
    target->resize(rows(), cols());

    Pointf p; // initialize to zeros
    p.status = PointStatusFlag::Invalid;

    // TODO only in device?
    target->fill(p);

    const auto& camera_matrix   = sensor_->cameraMatrix();
    const float ifx             = 1.f / camera_matrix(0, 0);
    const float ify             = 1.f / camera_matrix(1, 1);
    const float cx              = camera_matrix(0, 2);
    const float cy              = camera_matrix(1, 2);
    const Eigen::Matrix3f inv_K = camera_matrix.inverse();

    // init bidimensional kernel since we move in image space
    dim3 n_blocks(16, 16);
    dim3 n_threads;
    n_threads.x = (cols() + n_blocks.x - 1) / n_blocks.x;
    n_threads.y = (rows() + n_blocks.y - 1) / n_blocks.y;

    toCloud_kernel<<<n_blocks, n_threads>>>(target->deviceInstance(),
                                            matrix_.deviceInstance(),
                                            sensor_->sensorOffset(),
                                            inv_K,
                                            sensor_->cameraType(),
                                            ifx,
                                            ify,
                                            cx,
                                            cy,
                                            attributes_->min_depth,
                                            attributes_->max_depth);
    CUDA_CHECK(hipDeviceSynchronize());
  }

  void ImageSet::fromCloud(const MatrixCloud& src_cloud) {
    ImageEntry zero_entry;
    zero_entry.setDepth(attributes_->max_depth + 1);
    matrix_.fill(zero_entry);

    Eigen::Isometry3f inv_sensor_offset = sensor_->sensorOffset().inverse();
    Eigen::Vector3f sperical_point;
    Eigen::Vector3f coordinates;
    Eigen::Vector3f camera_point = Eigen::Vector3f::Zero();
    const float fx               = sensor_->cameraMatrix()(0, 0);
    const float fy               = sensor_->cameraMatrix()(1, 1);
    const float cx               = sensor_->cameraMatrix()(0, 2);
    const float cy               = sensor_->cameraMatrix()(1, 2);
    float w                      = 0;

    for (size_t i = 0; i < src_cloud.size(); ++i) {
      auto& src = src_cloud[i];
      if (src.status != PointStatusFlag::Good)
        continue;
      coordinates   = inv_sensor_offset * src.coordinates;
      const float x = coordinates.x();
      const float y = coordinates.y();
      const float z = coordinates.z();
      switch (sensor_->cameraType()) {
        case Pinhole: {
          w = coordinates(2);
          if (w < attributes_->min_depth || w > attributes_->max_depth)
            continue;
          camera_point = sensor_->cameraMatrix() * coordinates;
          camera_point.block<2, 1>(0, 0) *= 1. / w;
        } break;
        case Spherical: {
          w = coordinates.norm();
          if (w < attributes_->min_depth || w > attributes_->max_depth)
            continue;
          sperical_point.x() = atan2(y, x);
          sperical_point.y() = atan2(coordinates.z(), sqrt(x * x + y * y));
          sperical_point.z() = z;
          camera_point.x()   = fx * sperical_point.x() + cx;
          camera_point.y()   = fy * sperical_point.y() + cy;
          camera_point.z()   = w;
        } break;
        default:;
      }

      int c = (int) (camera_point.y() + (camera_point.y() >= 0 ? 0.5f : -0.5f));
      int r = (int) (camera_point.x() + (camera_point.x() >= 0 ? 0.5f : -0.5f));

      if (!matrix_.inside(r, c))
        continue;
      ImageEntry& entry = matrix_.at(r, c);

      if (w < entry.depth()) {
        entry.setIntensity(src.intensity);
        entry.setDepth(w);
        entry.setNormal(inv_sensor_offset.linear() * src.normal);
#ifdef _MD_ENABLE_SUPERRES_
        entry.c = camera_point.x();
        entry.r = camera_point.y();
#endif
        entry.setMasked(false);
      }
    }

    growMask();
    updateDerivatives();
  }

  void ImageSet::toCloud(MatrixCloud& target) const {
    target.resize(rows(), cols());
    Pointf p;
    p.status = PointStatusFlag::Invalid;
    target.fill(p);

    const float ifx = 1. / sensor_->cameraMatrix()(0, 0);
    const float ify = 1. / sensor_->cameraMatrix()(1, 1);
    const float cx  = sensor_->cameraMatrix()(0, 2);
    const float cy  = sensor_->cameraMatrix()(1, 2);

    Eigen::Matrix3f inv_K = sensor_->cameraMatrix().inverse();
    for (int r = 0; r < rows(); ++r) {
      for (int c = 0; c < cols(); ++c) {
        const ImageEntry& src = matrix_.at(r, c);
        Pointf& dest          = target.at(r, c);
        float w               = src.depth();
        if (src.masked() || w < attributes_->min_depth || w > attributes_->max_depth)
          continue;
#ifdef _MD_ENABLE_SUPERRES_
        const float row = src.r;
        const float col = src.c;
#else
        const float row = r;
        const float col = c;
#endif
        dest.status = PointStatusFlag::Good;
        switch (sensor_->cameraType()) {
          case Pinhole: {
            Eigen::Vector3f p = inv_K * Eigen::Vector3f(col * w, row * w, w);
            dest.coordinates  = p;
          } break;
          case Spherical: {
            float azimuth       = ifx * (col - cx);
            float elevation     = ify * (row - cy);
            float s0            = sin(azimuth);
            float c0            = cos(azimuth);
            float s1            = sin(elevation);
            float c1            = cos(elevation);
            dest.coordinates(0) = c0 * c1 * w;
            dest.coordinates(1) = s0 * c1 * w;
            dest.coordinates(2) = s1 * w;
          } break;
          default:;
        }

        dest.intensity   = src.intensity();
        dest.coordinates = sensor_->sensorOffset() * dest.coordinates;
        dest.normal      = sensor_->sensorOffset().linear() * src.normal();
      }
    }
  }

} // namespace photo

namespace photo {
  template <typename Matrix_>
  __device__ void __applyPolicy(ImageEntry& entry, Matrix_&& m, FilterPolicy policy, float squared_threshold) {
    if (entry.masked())
      return;

    float n = m.squaredNorm();
    if (n < squared_threshold)
      return;

    switch (policy) {
      case Suppress:
        entry.setMasked(1);
        break;
      case Clamp:
        m *= sqrt(squared_threshold / n);
        break;
      default:;
    }
  }
  __global__ void updateDerivatives_kernel(Image* matrix_,
                                           const float threshold_intensity,
                                           const float threshold_depth,
                                           const float threshold_normal,
                                           const FilterPolicy policy_intensity,
                                           const FilterPolicy policy_depth,
                                           const FilterPolicy policy_normal) {
    const auto rows = matrix_->rows();
    const auto cols = matrix_->cols();
    int row         = threadIdx.y + blockIdx.y * blockDim.y;
    int col         = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < 1 || row >= rows - 1 || col < 1 || col >= cols - 1)
      return;

    ImageEntry& entry          = matrix_->at<1>(row, col);
    const ImageEntry& entry_r0 = matrix_->at<1>(row - 1, col);
    const ImageEntry& entry_r1 = matrix_->at<1>(row + 1, col);
    const ImageEntry& entry_c0 = matrix_->at<1>(row, col - 1);
    const ImageEntry& entry_c1 = matrix_->at<1>(row, col + 1);

    // retrieve value
    const Vector5f& v_r0 = entry_r0.value;
    const Vector5f& v_r1 = entry_r1.value;
    const Vector5f& v_c0 = entry_c0.value;
    const Vector5f& v_c1 = entry_c1.value;

    // comptue derivatives
    Matrix5_2f& derivatives = entry.derivatives;
    derivatives.col(1)      = .5 * v_r1 - .5 * v_r0;
    derivatives.col(0)      = .5 * v_c1 - .5 * v_c0;

    __applyPolicy(entry, derivatives.row(0), policy_intensity, threshold_intensity);
    __applyPolicy(entry, derivatives.row(1), policy_depth, threshold_depth);
    __applyPolicy(entry, derivatives.block<3, 2>(2, 0), policy_normal, threshold_normal);
  }

  // void ImageSet::updateDerivativesDevice(Image& matrix_,
  //                                        const float threshold_intensity,
  //                                        const float threshold_depth,
  //                                        const float threshold_normal,
  //                                        const FilterPolicy policy_intensity,
  //                                        const FilterPolicy policy_depth,
  //                                        const FilterPolicy policy_normal) {
  //   const float i2 = threshold_intensity * threshold_intensity;
  //   const float d2 = threshold_depth * threshold_depth;
  //   const float n2 = threshold_normal * threshold_normal;
  //   updateDerivatives_kernel<<<dim3((cols() + 16 - 1) / 16, (rows() + 16 - 1) / 16), dim3(16, 16)>>>(
  //     matrix_.deviceInstance(), i2, d2, n2, policy_intensity, policy_depth, policy_normal);
  // }
  void ImageSet::updateDerivativesDevice() {
    const float i2 = pow(attributes_->thresholds[Intensity], 2);
    const float d2 = pow(attributes_->thresholds[Depth], 2);
    const float n2 = pow(attributes_->thresholds[Normal], 2);

    updateDerivatives_kernel<<<dim3((cols() + 16 - 1) / 16, (rows() + 16 - 1) / 16), dim3(16, 16)>>>(
      matrix_.deviceInstance(),
      i2,
      d2,
      n2,
      attributes_->policies[Intensity],
      attributes_->policies[Depth],
      attributes_->policies[Normal]);
  }
  __global__ void __copy_mask_kernel(Image* matrix_, uint8_t* d_old_mask, const size_t height, const size_t width) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row >= height || col >= width)
      return;

    const auto v_mask = matrix_->at<1>(row, col).masked();

    d_old_mask[row * width + col] = v_mask;
  }

  __global__ void
  __grow_mask_kernel(Image* matrix_, uint8_t* d_old_mask, int* d_ball_offsets, const int num_offsets, const size_t lenght) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= lenght)
      return;

    if (d_old_mask[tid] == 0)
      return;

    for (int i = 0; i < num_offsets; ++i) {
      int target = tid + d_ball_offsets[i];
      if (target < 0 || target >= lenght)
        continue;
      matrix_->at<1>(target).setMasked(true);
    }
  }

  void ImageSet::growMaskDevice() {
    const int& radius = attributes_->mask_grow_radius;
    std::vector<int> ball_offsets;
    int r2 = pow(radius, 2);
    for (int r = -radius; r < radius + 1; ++r) {
      for (int c = -radius; c < radius + 1; ++c) {
        int idx = r * cols() + c;
        if ((r * r + c * c) <= r2) {
          ball_offsets.push_back(idx);
        }
      }
    }
    const int num_offsets = static_cast<int>(ball_offsets.size());
    int* d_ball_offsets   = nullptr;
    CUDA_CHECK(hipMalloc(&d_ball_offsets, num_offsets * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_ball_offsets, ball_offsets.data(), num_offsets * sizeof(int), hipMemcpyHostToDevice));

    const size_t height = rows();
    const size_t width  = cols();
    uint8_t* d_old_mask;
    CUDA_CHECK(hipMalloc(&d_old_mask, height * width * sizeof(uint8_t)));
    // Launch copy kernel to copy the old mask
    __copy_mask_kernel<<<dim3((width + 16 - 1) / 16, (height + 16 - 1) / 16), dim3(16, 16)>>>(
      matrix_.deviceInstance(), d_old_mask, height, width);
    CUDA_CHECK(hipDeviceSynchronize());
    // Launch the kernel to grow the mask
    const size_t num_items = height * width;
    __grow_mask_kernel<<<(num_items + 1024 - 1) / 1024, 1024>>>(
      matrix_.deviceInstance(), d_old_mask, d_ball_offsets, num_offsets, num_items);
  }
} // namespace photo
